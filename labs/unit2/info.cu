#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include "gpuErrchk.h"

using namespace std;


int main(int argc, char **argv)
{
    // Get number of devices on system
    int deviceCount; 
    gpuErrchk(hipGetDeviceCount(&deviceCount));

    cout << "Number of devices: " << deviceCount << endl;
    for (int i = 0; i < deviceCount; ++i) 
    {
        // Get properties for device
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        cout << "Device " << i << endl;
        cout << "Name " << deviceProp.name << endl;
        cout << "Revision " << deviceProp.major << "." << deviceProp.minor << endl;
        cout << "Memory " << deviceProp.totalGlobalMem / 1024 / 1024 << "MB" << endl;
        cout << "Warp Size " << deviceProp.warpSize << endl;
        
        // Get clock rates using the new API (CUDA 13.0+)
        int clockRateKHz;
        hipDeviceGetAttribute(&clockRateKHz, hipDeviceAttributeClockRate, 0);
        cout << "Clock " << clockRateKHz / 1e3f << " Mhz" << endl;

        cout << "Multiprocessors " << deviceProp.multiProcessorCount << endl;
    } 
    return 0;
}